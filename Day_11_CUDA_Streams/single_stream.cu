
#include <hip/hip_runtime.h>
#include <stdio.h>


#define N (1024*1024)
#define FULL_DATA_SIZE (N*200)


static void
HandleError (hipError_t err, const char *file, int line)
{
  if (err != hipSuccess)
    {
      printf ("%s in %s at line %d\n", hipGetErrorString (err), file, line);
      exit (EXIT_FAILURE);
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



__global__ void
kernel (int *a, int *b, int *c)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N)
    {
      int idx1 = (idx + 1) % 256;
      int idx2 = (idx + 2) % 256;
      float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
      float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
      c[idx] = (as + bs) / 2;
    }
}


int
main ()
{

  hipEvent_t start, stop;
  float elapsedTime;
  // start the timers
  HANDLE_ERROR (hipEventCreate (&start));
  HANDLE_ERROR (hipEventCreate (&stop));
  HANDLE_ERROR (hipEventRecord (start, 0));

  // initialize the stream
  hipStream_t stream;
  HANDLE_ERROR (hipStreamCreate (&stream));

  int *host_a, *host_b, *host_c;
  int *dev_a, *dev_b, *dev_c;

  // allocate the memory on the GPU
  HANDLE_ERROR (hipMalloc ((void **) &dev_a, N * sizeof (int)));
  HANDLE_ERROR (hipMalloc ((void **) &dev_b, N * sizeof (int)));
  HANDLE_ERROR (hipMalloc ((void **) &dev_c, N * sizeof (int)));

  // allocate page-locked memory, used to stream
  HANDLE_ERROR (hipHostAlloc
		((void **) &host_a, FULL_DATA_SIZE * sizeof (int),
		 hipHostMallocDefault));
  HANDLE_ERROR (hipHostAlloc
		((void **) &host_b, FULL_DATA_SIZE * sizeof (int),
		 hipHostMallocDefault));
  HANDLE_ERROR (hipHostAlloc
		((void **) &host_c, FULL_DATA_SIZE * sizeof (int),
		 hipHostMallocDefault));

  for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
      host_a[i] = rand ();
      host_b[i] = rand ();
    }

  // now loop over full data, in bite-sized chunks
  for (int i = 0; i < FULL_DATA_SIZE; i += N)
    {
      // copy the locked memory to the device, async
      HANDLE_ERROR (hipMemcpyAsync
		    (dev_a, host_a + i, N * sizeof (int),
		     hipMemcpyHostToDevice, stream));
      HANDLE_ERROR (hipMemcpyAsync
		    (dev_b, host_b + i, N * sizeof (int),
		     hipMemcpyHostToDevice, stream));
      kernel <<< N / 256, 256, 0, stream >>> (dev_a, dev_b, dev_c);
      // copy the data from device to locked memory
      HANDLE_ERROR (hipMemcpyAsync
		    (host_c + i, dev_c, N * sizeof (int),
		     hipMemcpyDeviceToHost, stream));
    }

  // copy result chunk from locked to full buffer
  HANDLE_ERROR (hipStreamSynchronize (stream));

  HANDLE_ERROR (hipEventRecord (stop, 0));
  HANDLE_ERROR (hipEventSynchronize (stop));
  HANDLE_ERROR (hipEventElapsedTime (&elapsedTime, start, stop));
  printf ("Time taken:  %3.1f ms\n", elapsedTime);
  // cleanup the streams and memory
  HANDLE_ERROR (hipHostFree (host_a));
  HANDLE_ERROR (hipHostFree (host_b));
  HANDLE_ERROR (hipHostFree (host_c));
  HANDLE_ERROR (hipFree (dev_a));
  HANDLE_ERROR (hipFree (dev_b));
  HANDLE_ERROR (hipFree (dev_c));
  HANDLE_ERROR (hipStreamDestroy (stream));
  return 0;

}