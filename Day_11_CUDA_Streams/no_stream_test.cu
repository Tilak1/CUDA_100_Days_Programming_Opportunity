// no_streams.cu
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void dummy_kernel(float* data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        data[idx] = data[idx] * 2.0f;
}

int main() {
    int N = 1 << 24;
    size_t size = N * sizeof(float);
    float *h_data = new float[N];
    float *d_data;

    hipMalloc(&d_data, size);

    for (int i = 0; i < N; ++i) h_data[i] = 1.0f;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Blocking copy
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

    dummy_kernel<<<(N + 255) / 256, 256>>>(d_data, N);
    hipDeviceSynchronize();

    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Time without streams: " << ms << " ms\n";

    hipFree(d_data);
    delete[] h_data;
    return 0;
}
