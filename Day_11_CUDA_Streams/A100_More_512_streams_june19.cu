#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)
#define STREAMS_PER_GPU 512  // Test high stream counts: 32, 64, 128, 256, 512

static void
HandleError (hipError_t err, const char *file, int line)
{
  if (err != hipSuccess)
    {
      printf ("%s in %s at line %d\n", hipGetErrorString (err), file, line);
      exit (EXIT_FAILURE);
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void
kernel (int *a, int *b, int *c)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N)
    {
      int idx1 = (idx + 1) % 256;
      int idx2 = (idx + 2) % 256;
      float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
      float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
      c[idx] = (as + bs) / 2;
    }
}

int
main (void)
{
  int deviceCount;
  HANDLE_ERROR(hipGetDeviceCount(&deviceCount));
  
  if (deviceCount < 1) {
    printf("No CUDA devices found!\n");
    return 1;
  }
  
  printf("=== A100 High-Stream Multi-GPU Analysis ===\n");
  printf("Found %d CUDA device(s)\n", deviceCount);
  printf("Streams per GPU: %d (testing high concurrency)\n", STREAMS_PER_GPU);
  printf("Total streams: %d\n", deviceCount * STREAMS_PER_GPU);
  printf("Data size: %ld MB per GPU\n", (FULL_DATA_SIZE * sizeof(int) * 3) / (1024 * 1024));
  printf("Total data size: %ld MB\n", (FULL_DATA_SIZE * sizeof(int) * 3) / (1024 * 1024));
  
  // Check device properties and enable peer access
  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));
    printf("Device %d: %s\n", dev, prop.name);
    printf("  Memory: %.1f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  Concurrent Kernels: %s\n", prop.concurrentKernels ? "YES" : "NO");
    
    if (!prop.deviceOverlap) {
      printf("  WARNING: Device %d will not handle overlaps!\n", dev);
    }
  }
  
  // Enable peer-to-peer access between GPUs (if available)
  printf("\nChecking P2P connectivity:\n");
  for (int dev1 = 0; dev1 < deviceCount; dev1++) {
    for (int dev2 = 0; dev2 < deviceCount; dev2++) {
      if (dev1 != dev2) {
        int canAccessPeer;
        HANDLE_ERROR(hipDeviceCanAccessPeer(&canAccessPeer, dev1, dev2));
        if (canAccessPeer) {
          HANDLE_ERROR(hipSetDevice(dev1));
          hipError_t err = hipDeviceEnablePeerAccess(dev2, 0);
          if (err == hipSuccess) {
            printf("  P2P enabled: GPU %d <-> GPU %d\n", dev1, dev2);
          } else if (err != hipErrorPeerAccessAlreadyEnabled) {
            printf("  P2P failed: GPU %d <-> GPU %d\n", dev1, dev2);
          }
        }
      }
    }
  }
  printf("\n");

  // Timing events
  hipEvent_t *start_per_gpu = (hipEvent_t*)malloc(deviceCount * sizeof(hipEvent_t));
  hipEvent_t *stop_per_gpu = (hipEvent_t*)malloc(deviceCount * sizeof(hipEvent_t));
  hipEvent_t global_start, global_stop;
  hipEvent_t launch_start, launch_stop;  // For measuring pure launch time
  hipEvent_t kernel_launch_start, kernel_launch_stop;  // For pure kernel launch overhead
  hipEvent_t memcpy_start, memcpy_stop;  // For memory transfer timing
  
  // Create timing events for each GPU
  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    HANDLE_ERROR(hipEventCreate(&start_per_gpu[dev]));
    HANDLE_ERROR(hipEventCreate(&stop_per_gpu[dev]));
  }
  
  // Global timing events on device 0
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventCreate(&global_start));
  HANDLE_ERROR(hipEventCreate(&global_stop));
  HANDLE_ERROR(hipEventCreate(&launch_start));
  HANDLE_ERROR(hipEventCreate(&launch_stop));
  HANDLE_ERROR(hipEventCreate(&kernel_launch_start));
  HANDLE_ERROR(hipEventCreate(&kernel_launch_stop));
  HANDLE_ERROR(hipEventCreate(&memcpy_start));
  HANDLE_ERROR(hipEventCreate(&memcpy_stop));

  // Arrays to hold streams and device pointers for each GPU
  hipStream_t *streams = (hipStream_t*)malloc(deviceCount * STREAMS_PER_GPU * sizeof(hipStream_t));
  int **dev_a = (int**)malloc(deviceCount * STREAMS_PER_GPU * sizeof(int*));
  int **dev_b = (int**)malloc(deviceCount * STREAMS_PER_GPU * sizeof(int*));
  int **dev_c = (int**)malloc(deviceCount * STREAMS_PER_GPU * sizeof(int*));

  // Initialize streams and allocate memory on each GPU
  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    
    // Create STREAMS_PER_GPU streams per GPU
    for (int s = 0; s < STREAMS_PER_GPU; s++) {
      HANDLE_ERROR(hipStreamCreate(&streams[dev * STREAMS_PER_GPU + s]));
      
      // Allocate GPU memory buffers for each stream
      HANDLE_ERROR(hipMalloc((void**)&dev_a[dev * STREAMS_PER_GPU + s], N * sizeof(int)));
      HANDLE_ERROR(hipMalloc((void**)&dev_b[dev * STREAMS_PER_GPU + s], N * sizeof(int)));
      HANDLE_ERROR(hipMalloc((void**)&dev_c[dev * STREAMS_PER_GPU + s], N * sizeof(int)));
    }
  }

  // Allocate page-locked host memory
  int *host_a, *host_b, *host_c;
  HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

  // Initialize host data
  printf("Initializing data...\n");
  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = rand();
    host_b[i] = rand();
  }

  // Calculate work distribution
  int totalChunks = FULL_DATA_SIZE / N;
  int chunksPerGpu = totalChunks / deviceCount;
  int remainingChunks = totalChunks % deviceCount;
  
  printf("Work distribution:\n");
  printf("  Total chunks: %d\n", totalChunks);
  printf("  Chunks per GPU: %d\n", chunksPerGpu);
  printf("  Remaining chunks: %d\n", remainingChunks);
  printf("  Streams per GPU: %d\n", STREAMS_PER_GPU);
  printf("  Total streams: %d\n", deviceCount * STREAMS_PER_GPU);
  
  if (chunksPerGpu >= STREAMS_PER_GPU) {
    printf("  Chunks per stream: ~%d\n", chunksPerGpu / STREAMS_PER_GPU);
    printf("  Stream utilization: FULL\n");
  } else {
    printf("  Chunks per stream: <1 (some streams idle)\n");
    printf("  Stream utilization: PARTIAL (%d/%d streams active per GPU)\n", chunksPerGpu, STREAMS_PER_GPU);
  }
  
  printf("  Memory per stream: ~%ld KB\n", (N * sizeof(int) * 3) / 1024);
  printf("\n");

  // ==============================================
  // SEQUENTIAL MULTI-GPU (for comparison) - using only 2 streams
  // ==============================================
  printf("=== Sequential Multi-GPU Launch (2 streams baseline) ===\n");
  
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(global_start, 0));
  
  // Sequential approach - launch kernels one GPU at a time (using first 2 streams only)
  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    HANDLE_ERROR(hipEventRecord(start_per_gpu[dev], 0));
    
    int startChunk = dev * chunksPerGpu + (dev < remainingChunks ? dev : remainingChunks);
    int endChunk = startChunk + chunksPerGpu + (dev < remainingChunks ? 1 : 0);
    
    for (int chunk = startChunk; chunk < endChunk; chunk += 2) {
      int dataOffset1 = chunk * N;
      int dataOffset2 = (chunk + 1) * N;
      
      // Stream 0
      if (chunk < endChunk) {
        HANDLE_ERROR(hipMemcpyAsync(dev_a[dev * STREAMS_PER_GPU], host_a + dataOffset1, 
                     N * sizeof(int), hipMemcpyHostToDevice, streams[dev * STREAMS_PER_GPU]));
        HANDLE_ERROR(hipMemcpyAsync(dev_b[dev * STREAMS_PER_GPU], host_b + dataOffset1, 
                     N * sizeof(int), hipMemcpyHostToDevice, streams[dev * STREAMS_PER_GPU]));
        kernel<<<N / 256, 256, 0, streams[dev * STREAMS_PER_GPU]>>>(dev_a[dev * STREAMS_PER_GPU], dev_b[dev * STREAMS_PER_GPU], dev_c[dev * STREAMS_PER_GPU]);
        HANDLE_ERROR(hipMemcpyAsync(host_c + dataOffset1, dev_c[dev * STREAMS_PER_GPU], 
                     N * sizeof(int), hipMemcpyDeviceToHost, streams[dev * STREAMS_PER_GPU]));
      }
      
      // Stream 1
      if (chunk + 1 < endChunk) {
        HANDLE_ERROR(hipMemcpyAsync(dev_a[dev * STREAMS_PER_GPU + 1], host_a + dataOffset2, 
                     N * sizeof(int), hipMemcpyHostToDevice, streams[dev * STREAMS_PER_GPU + 1]));
        HANDLE_ERROR(hipMemcpyAsync(dev_b[dev * STREAMS_PER_GPU + 1], host_b + dataOffset2, 
                     N * sizeof(int), hipMemcpyHostToDevice, streams[dev * STREAMS_PER_GPU + 1]));
        kernel<<<N / 256, 256, 0, streams[dev * STREAMS_PER_GPU + 1]>>>(dev_a[dev * STREAMS_PER_GPU + 1], dev_b[dev * STREAMS_PER_GPU + 1], dev_c[dev * STREAMS_PER_GPU + 1]);
        HANDLE_ERROR(hipMemcpyAsync(host_c + dataOffset2, dev_c[dev * STREAMS_PER_GPU + 1], 
                     N * sizeof(int), hipMemcpyDeviceToHost, streams[dev * STREAMS_PER_GPU + 1]));
      }
    }
    
    HANDLE_ERROR(hipStreamSynchronize(streams[dev * STREAMS_PER_GPU]));
    HANDLE_ERROR(hipStreamSynchronize(streams[dev * STREAMS_PER_GPU + 1]));
    HANDLE_ERROR(hipEventRecord(stop_per_gpu[dev], 0));
  }
  
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(global_stop, 0));
  HANDLE_ERROR(hipEventSynchronize(global_stop));
  
  float sequentialTime;
  HANDLE_ERROR(hipEventElapsedTime(&sequentialTime, global_start, global_stop));
  printf("Sequential multi-GPU time: %8.1f ms\n\n", sequentialTime);

  // Reset result array
  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_c[i] = 0;
  }

  // ==============================================
  // SIMULTANEOUS MULTI-GPU LAUNCH WITH MULTIPLE STREAMS
  // ==============================================  
  printf("=== Simultaneous Multi-GPU Launch with %d Streams per GPU ===\n", STREAMS_PER_GPU);
  
  // Pre-copy all data to GPUs first to isolate kernel launch overhead
  printf("Pre-copying data to all GPUs (%d streams per GPU)...\n", STREAMS_PER_GPU);
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(memcpy_start, 0));
  
  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    
    int startChunk = dev * chunksPerGpu + (dev < remainingChunks ? dev : remainingChunks);
    int endChunk = startChunk + chunksPerGpu + (dev < remainingChunks ? 1 : 0);
    int chunksForThisGpu = endChunk - startChunk;
    
    // Distribute chunks across all streams for this GPU
    for (int chunk = startChunk; chunk < endChunk; chunk++) {
      int streamIdx = (chunk - startChunk) % STREAMS_PER_GPU;
      int bufferIdx = dev * STREAMS_PER_GPU + streamIdx;
      int dataOffset = chunk * N;
      
      // Copy data for this stream
      HANDLE_ERROR(hipMemcpy(dev_a[bufferIdx], host_a + dataOffset, 
                 N * sizeof(int), hipMemcpyHostToDevice));
      HANDLE_ERROR(hipMemcpy(dev_b[bufferIdx], host_b + dataOffset, 
                 N * sizeof(int), hipMemcpyHostToDevice));
    }
    HANDLE_ERROR(hipDeviceSynchronize());
  }
  
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(memcpy_stop, 0));
  HANDLE_ERROR(hipEventSynchronize(memcpy_stop));
  
  float memcpyTime;
  HANDLE_ERROR(hipEventElapsedTime(&memcpyTime, memcpy_start, memcpy_stop));
  printf("Data pre-copy time: %8.1f ms\n", memcpyTime);
  
  // Now measure PURE KERNEL LAUNCH overhead AND per-GPU execution
  printf("\nMeasuring pure kernel launch overhead with %d streams per GPU...\n", STREAMS_PER_GPU);
  
  // Start per-GPU timing BEFORE launching kernels
  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    HANDLE_ERROR(hipEventRecord(start_per_gpu[dev], 0));
  }
  
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(kernel_launch_start, 0));
  
  // Launch ALL kernels across ALL GPUs and ALL streams
  int totalKernelsLaunched = 0;
  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    
    int startChunk = dev * chunksPerGpu + (dev < remainingChunks ? dev : remainingChunks);
    int endChunk = startChunk + chunksPerGpu + (dev < remainingChunks ? 1 : 0);
    int chunksForThisGpu = endChunk - startChunk;
    
    printf("  GPU %d: launching %d kernels across %d streams\n", dev, chunksForThisGpu, STREAMS_PER_GPU);
    
    // Launch kernels distributed across all streams
    for (int chunk = startChunk; chunk < endChunk; chunk++) {
      int streamIdx = (chunk - startChunk) % STREAMS_PER_GPU;
      int bufferIdx = dev * STREAMS_PER_GPU + streamIdx;
      int streamGlobalIdx = dev * STREAMS_PER_GPU + streamIdx;
      
      kernel<<<N / 256, 256, 0, streams[streamGlobalIdx]>>>(dev_a[bufferIdx], dev_b[bufferIdx], dev_c[bufferIdx]);
      totalKernelsLaunched++;
    }
  }
  
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(kernel_launch_stop, 0));
  HANDLE_ERROR(hipEventSynchronize(kernel_launch_stop));
  
  float pureKernelLaunchTime;
  HANDLE_ERROR(hipEventElapsedTime(&pureKernelLaunchTime, kernel_launch_start, kernel_launch_stop));
  
  printf("Pure kernel launch time: %6.3f ms (%d kernels)\n", pureKernelLaunchTime, totalKernelsLaunched);
  
  // Wait for all kernels to complete and measure execution time
  printf("Waiting for kernel execution to complete...\n");
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(global_start, 0));
  
  float *gpuTimes = (float*)malloc(deviceCount * sizeof(float));
  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    
    // Synchronize all streams for this GPU
    for (int s = 0; s < STREAMS_PER_GPU; s++) {
      HANDLE_ERROR(hipStreamSynchronize(streams[dev * STREAMS_PER_GPU + s]));
    }
    
    HANDLE_ERROR(hipEventRecord(stop_per_gpu[dev], 0));
    HANDLE_ERROR(hipEventSynchronize(stop_per_gpu[dev]));
    HANDLE_ERROR(hipEventElapsedTime(&gpuTimes[dev], start_per_gpu[dev], stop_per_gpu[dev]));
  }
  
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(global_stop, 0));
  HANDLE_ERROR(hipEventSynchronize(global_stop));
  
  float kernelExecutionTime;
  HANDLE_ERROR(hipEventElapsedTime(&kernelExecutionTime, global_start, global_stop));
  
  // Copy results back to host using all streams
  printf("Copying results back to host using %d streams per GPU...\n", STREAMS_PER_GPU);
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(memcpy_start, 0));
  
  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    
    int startChunk = dev * chunksPerGpu + (dev < remainingChunks ? dev : remainingChunks);
    int endChunk = startChunk + chunksPerGpu + (dev < remainingChunks ? 1 : 0);
    
    for (int chunk = startChunk; chunk < endChunk; chunk++) {
      int streamIdx = (chunk - startChunk) % STREAMS_PER_GPU;
      int bufferIdx = dev * STREAMS_PER_GPU + streamIdx;
      int dataOffset = chunk * N;
      
      HANDLE_ERROR(hipMemcpy(host_c + dataOffset, dev_c[bufferIdx], 
                 N * sizeof(int), hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipDeviceSynchronize());
  }
  
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipEventRecord(memcpy_stop, 0));
  HANDLE_ERROR(hipEventSynchronize(memcpy_stop));
  
  float resultCopyTime;
  HANDLE_ERROR(hipEventElapsedTime(&resultCopyTime, memcpy_start, memcpy_stop));
  
  float totalTimeDetailed = memcpyTime + pureKernelLaunchTime + kernelExecutionTime + resultCopyTime;

  // ==============================================
  // DETAILED RESULTS ANALYSIS
  // ==============================================
  printf("\n=== Detailed Performance Breakdown ===\n");
  printf("Sequential multi-GPU:    %8.1f ms\n", sequentialTime);
  printf("\nDetailed simultaneous timing:\n");
  printf("1. Data H2D copy:        %8.1f ms\n", memcpyTime);
  printf("2. Pure kernel launch:   %8.3f ms  ⭐ PURE OVERHEAD\n", pureKernelLaunchTime);
  printf("3. Kernel execution:     %8.1f ms\n", kernelExecutionTime);
  printf("4. Result D2H copy:      %8.1f ms\n", resultCopyTime);
  printf("   Total detailed:       %8.1f ms\n", totalTimeDetailed);
  
  printf("\nPer-GPU execution times:\n");
  float maxGpuTime = 0;
  float minGpuTime = gpuTimes[0];
  float avgGpuTime = 0;
  
  for (int dev = 0; dev < deviceCount; dev++) {
    printf("  GPU %d execution:      %8.1f ms\n", dev, gpuTimes[dev]);
    maxGpuTime = (gpuTimes[dev] > maxGpuTime) ? gpuTimes[dev] : maxGpuTime;
    minGpuTime = (gpuTimes[dev] < minGpuTime) ? gpuTimes[dev] : minGpuTime;
    avgGpuTime += gpuTimes[dev];
  }
  avgGpuTime /= deviceCount;
  
  printf("\n=== Pure Kernel Launch Analysis ===\n");
  int totalKernels = 0;
  for (int dev = 0; dev < deviceCount; dev++) {
    int startChunk = dev * chunksPerGpu + (dev < remainingChunks ? dev : remainingChunks);
    int endChunk = startChunk + chunksPerGpu + (dev < remainingChunks ? 1 : 0);
    totalKernels += ((endChunk - startChunk + 1) / 2) * 2;
  }
  
  float launchOverheadPerKernel = pureKernelLaunchTime / totalKernels;
  float launchOverheadPerGpu = pureKernelLaunchTime / deviceCount;
  
  printf("Total kernels launched:  %8d\n", totalKernels);
  printf("Launch overhead per kernel: %5.1f μs\n", launchOverheadPerKernel * 1000);
  printf("Launch overhead per GPU:    %5.1f μs\n", launchOverheadPerGpu * 1000);
  printf("Kernels per GPU:            %5d\n", totalKernels / deviceCount);
  
  printf("\n=== Parallelism Analysis ===\n");
  float speedup = sequentialTime / totalTimeDetailed;
  float efficiency = speedup / deviceCount * 100;
  
  printf("Overall speedup:         %8.2fx\n", speedup);
  printf("Parallel efficiency:     %8.1f%%\n", efficiency);
  printf("GPU time variance:       %8.1f ms\n", maxGpuTime - minGpuTime);
  printf("Kernel execution sync:   %8.1f ms\n", kernelExecutionTime - maxGpuTime);
  
  // Launch overhead analysis
  if (pureKernelLaunchTime < 0.001) {
    printf("\n🚀 ULTRA-LOW kernel launch overhead (<1μs)!\n");
  } else if (pureKernelLaunchTime < 0.01) {
    printf("\n✓ EXCELLENT kernel launch overhead (<10μs)\n");
  } else if (pureKernelLaunchTime < 0.1) {
    printf("\n✓ GOOD kernel launch overhead (<100μs)\n");
  } else if (pureKernelLaunchTime < 1.0) {
    printf("\n⚠ MODERATE kernel launch overhead (<1ms)\n");
  } else {
    printf("\n❌ HIGH kernel launch overhead (>1ms)\n");
  }
  
  // Per-kernel launch overhead analysis
  if (launchOverheadPerKernel < 0.001) {
    printf("⚡ Per-kernel overhead: <1μs (EXCEPTIONAL)\n");
  } else if (launchOverheadPerKernel < 0.01) {
    printf("✓ Per-kernel overhead: <10μs (EXCELLENT)\n");
  } else if (launchOverheadPerKernel < 0.1) {
    printf("✓ Per-kernel overhead: <100μs (GOOD)\n");
  } else {
    printf("⚠ Per-kernel overhead: >100μs (HIGH)\n");
  }
  
  // Check if kernel execution is truly parallel
  if (kernelExecutionTime <= maxGpuTime * 1.05) {
    printf("✓ PERFECT kernel execution parallelism\n");
  } else {
    printf("⚠ Limited kernel execution parallelism\n");
  }
  
  printf("\n=== A100 Performance Summary ===\n");
  printf("Memory bandwidth utilization: %s\n", 
         (memcpyTime + resultCopyTime) < kernelExecutionTime ? "OPTIMAL" : "BOTTLENECK");
  printf("Compute utilization:          %s\n",
         kernelExecutionTime > (memcpyTime + resultCopyTime) ? "COMPUTE-BOUND" : "MEMORY-BOUND");
  printf("Multi-GPU scaling:            %s\n",
         efficiency > 85 ? "EXCELLENT" : efficiency > 70 ? "GOOD" : "LIMITED");

  // Cleanup
  HANDLE_ERROR(hipHostFree(host_a));
  HANDLE_ERROR(hipHostFree(host_b));
  HANDLE_ERROR(hipHostFree(host_c));

  for (int dev = 0; dev < deviceCount; dev++) {
    HANDLE_ERROR(hipSetDevice(dev));
    
    for (int s = 0; s < STREAMS_PER_GPU; s++) {
      HANDLE_ERROR(hipStreamDestroy(streams[dev * STREAMS_PER_GPU + s]));
      HANDLE_ERROR(hipFree(dev_a[dev * STREAMS_PER_GPU + s]));
      HANDLE_ERROR(hipFree(dev_b[dev * STREAMS_PER_GPU + s]));
      HANDLE_ERROR(hipFree(dev_c[dev * STREAMS_PER_GPU + s]));
    }
    
    HANDLE_ERROR(hipEventDestroy(start_per_gpu[dev]));
    HANDLE_ERROR(hipEventDestroy(stop_per_gpu[dev]));
  }

  free(streams);
  free(dev_a);
  free(dev_b);
  free(dev_c);
  free(start_per_gpu);
  free(stop_per_gpu);
  free(gpuTimes);

  HANDLE_ERROR(hipEventDestroy(global_start));
  HANDLE_ERROR(hipEventDestroy(global_stop));
  HANDLE_ERROR(hipEventDestroy(launch_start));
  HANDLE_ERROR(hipEventDestroy(launch_stop));
  HANDLE_ERROR(hipEventDestroy(kernel_launch_start));
  HANDLE_ERROR(hipEventDestroy(kernel_launch_stop));
  HANDLE_ERROR(hipEventDestroy(memcpy_start));
  HANDLE_ERROR(hipEventDestroy(memcpy_stop));

  return 0;
}