#include "hip/hip_runtime.h"
// Optimized GPU code for 64000 x 20 x 4096 pointwise multiply using double-buffered streams
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <vector>
#include <random>
#include <chrono>

constexpr int RAYS = 64000;
constexpr int FRAMES = 20;
constexpr int N = 4096; // FFT size
constexpr int TOTAL = RAYS * FRAMES * N;
constexpr int BLOCKSIZE = 256;
constexpr int CHUNK = TOTAL / 2; // Use two buffers

__global__ void flat_pw_multiply(const hipFloatComplex* X,
                                 const hipFloatComplex* H,
                                 hipFloatComplex* Y,
                                 int total) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < total) {
        Y[gid] = hipCmulf(X[gid], H[gid]);
    }
}

std::vector<hipFloatComplex> randomComplexVector(int count) {
    std::mt19937 gen(0xC0FFEE);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    std::vector<hipFloatComplex> vec(count);
    for (auto& c : vec) c = make_hipFloatComplex(dist(gen), dist(gen));
    return vec;
}

int main() {
    size_t chunk_bytes = CHUNK * sizeof(hipFloatComplex);

    // Allocate host pinned memory (double buffer)
    hipFloatComplex *h_X[2], *h_H[2], *h_Y[2];
    for (int i = 0; i < 2; ++i) {
        hipHostAlloc(&h_X[i], chunk_bytes, hipHostMallocDefault);
        hipHostAlloc(&h_H[i], chunk_bytes, hipHostMallocDefault);
        hipHostAlloc(&h_Y[i], chunk_bytes, hipHostMallocDefault);
    }

    // Fill input with dummy data
    auto rand_X = randomComplexVector(TOTAL);
    auto rand_H = randomComplexVector(TOTAL);
    memcpy(h_X[0], rand_X.data(), chunk_bytes);
    memcpy(h_H[0], rand_H.data(), chunk_bytes);
    memcpy(h_X[1], rand_X.data() + CHUNK, chunk_bytes);
    memcpy(h_H[1], rand_H.data() + CHUNK, chunk_bytes);

    // Allocate device memory (double buffer)
    hipFloatComplex *d_X[2], *d_H[2], *d_Y[2];
    for (int i = 0; i < 2; ++i) {
        hipMalloc(&d_X[i], chunk_bytes);
        hipMalloc(&d_H[i], chunk_bytes);
        hipMalloc(&d_Y[i], chunk_bytes);
    }

    // Create two streams
    hipStream_t streams[2];
    for (int i = 0; i < 2; ++i) hipStreamCreate(&streams[i]);

    // Timing start
    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < 2; ++i) {
        // Async H2D copy
        hipMemcpyAsync(d_X[i], h_X[i], chunk_bytes, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_H[i], h_H[i], chunk_bytes, hipMemcpyHostToDevice, streams[i]);

        // Kernel launch
        dim3 block(BLOCKSIZE);
        dim3 grid((CHUNK + block.x - 1) / block.x);
        flat_pw_multiply<<<grid, block, 0, streams[i]>>>(d_X[i], d_H[i], d_Y[i], CHUNK);

        // Async D2H copy
        hipMemcpyAsync(h_Y[i], d_Y[i], chunk_bytes, hipMemcpyDeviceToHost, streams[i]);
    }

    // Sync both streams
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);

    auto end = std::chrono::high_resolution_clock::now();
    double total_us = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    printf("Completed 64000x20x4096 in %.2f ms with double-buffered streams\n", total_us / 1000.0);

    // Cleanup
    for (int i = 0; i < 2; ++i) {
        hipFree(d_X[i]); hipFree(d_H[i]); hipFree(d_Y[i]);
        hipHostFree(h_X[i]); hipHostFree(h_H[i]); hipHostFree(h_Y[i]);
        hipStreamDestroy(streams[i]);
    }

    return 0;
}
