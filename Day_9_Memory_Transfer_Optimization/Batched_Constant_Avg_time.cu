// Optimized CUDA code using pinned memory and streams for pointwise multiply benchmarking
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <random>
#include <vector>

constexpr int N = 4096;
constexpr int BLOCKSIZE = 128;
constexpr int BATCHES = 1000;

__global__ void pw_const_kernel(const hipFloatComplex* __restrict__ X,
                                hipFloatComplex*       __restrict__ Y,
                                int                   n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) Y[i] = hipCmulf(X[i], Y[i]);
}

__constant__ hipFloatComplex H_const[N];

void gpuCheck(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error %s : %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

std::vector<hipFloatComplex> randomComplexVector(int n) {
    std::mt19937 gen(0xC0FFEE);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    std::vector<hipFloatComplex> v(n);
    for (auto& c : v) c = make_hipFloatComplex(dist(gen), dist(gen));
    return v;
}

int main() {
    dim3 block(BLOCKSIZE);
    dim3 grid((N + BLOCKSIZE - 1) / BLOCKSIZE);

    hipFloatComplex *d_X, *d_Y;
    hipFloatComplex *h_X_pinned, *h_Y_pinned;
    size_t bytes = N * sizeof(hipFloatComplex);

    // Allocate pinned host memory
    gpuCheck(hipHostAlloc(&h_X_pinned, bytes, hipHostMallocDefault), "host X pinned");
    gpuCheck(hipHostAlloc(&h_Y_pinned, bytes, hipHostMallocDefault), "host Y pinned");

    // Fill input data
    auto h_X_data = randomComplexVector(N);
    auto h_H_data = randomComplexVector(N);
    memcpy(h_X_pinned, h_X_data.data(), bytes);

    // Allocate device memory
    gpuCheck(hipMalloc(&d_X, bytes), "malloc d_X");
    gpuCheck(hipMalloc(&d_Y, bytes), "malloc d_Y");

    // Copy H to constant memory
    gpuCheck(hipMemcpyToSymbol(HIP_SYMBOL(H_const), h_H_data.data(), bytes), "const copy");

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Variables to accumulate timing data
    double total_h2d_time = 0.0;
    double total_kernel_time = 0.0;
    double total_d2h_time = 0.0;

    for (int b = 0; b < BATCHES; ++b) {
        hipEvent_t start_h2d, stop_h2d, start_kernel, stop_kernel, start_d2h, stop_d2h;
        hipEventCreate(&start_h2d);
        hipEventCreate(&stop_h2d);
        hipEventCreate(&start_kernel);
        hipEventCreate(&stop_kernel);
        hipEventCreate(&start_d2h);
        hipEventCreate(&stop_d2h);

        // Async H2D
        hipEventRecord(start_h2d, stream);
        hipMemcpyAsync(d_X, h_X_pinned, bytes, hipMemcpyHostToDevice, stream);
        hipEventRecord(stop_h2d, stream);

        // Kernel
        hipEventRecord(start_kernel, stream);
        pw_const_kernel<<<grid, block, 0, stream>>>(d_X, d_Y, N);
        hipEventRecord(stop_kernel, stream);

        // Async D2H
        hipEventRecord(start_d2h, stream);
        hipMemcpyAsync(h_Y_pinned, d_Y, bytes, hipMemcpyDeviceToHost, stream);
        hipEventRecord(stop_d2h, stream);

        // Sync and time
        hipEventSynchronize(stop_d2h);

        float h2d_time = 0.f, kernel_time = 0.f, d2h_time = 0.f;
        hipEventElapsedTime(&h2d_time, start_h2d, stop_h2d);
        hipEventElapsedTime(&kernel_time, start_kernel, stop_kernel);
        hipEventElapsedTime(&d2h_time, start_d2h, stop_d2h);

        // Accumulate times (convert ms to us)
        total_h2d_time += h2d_time * 1000.0;
        total_kernel_time += kernel_time * 1000.0;
        total_d2h_time += d2h_time * 1000.0;

        hipEventDestroy(start_h2d);
        hipEventDestroy(stop_h2d);
        hipEventDestroy(start_kernel);
        hipEventDestroy(stop_kernel);
        hipEventDestroy(start_d2h);
        hipEventDestroy(stop_d2h);
    }

    // Calculate and print averages
    double avg_h2d_time = total_h2d_time / BATCHES;
    double avg_kernel_time = total_kernel_time / BATCHES;
    double avg_d2h_time = total_d2h_time / BATCHES;

    //constexpr int BATCHES = 1000;
    printf("Benchmark Results (averaged over %d batches):\n", BATCHES);
    printf("Average H2D copy time: %.2f μs\n", avg_h2d_time);
    printf("Average kernel time: %.2f μs\n", avg_kernel_time);
    printf("Average D2H copy time: %.2f μs\n", avg_d2h_time);
    printf("Average total time per batch: %.2f μs\n", avg_h2d_time + avg_kernel_time + avg_d2h_time);

    hipFree(d_X); 
    hipFree(d_Y);
    hipHostFree(h_X_pinned); 
    hipHostFree(h_Y_pinned);
    hipStreamDestroy(stream);

    return 0;
}