#include <cstdio>
 #include <hip/hip_runtime.h>
 #include <hip/hip_complex.h>
 #include <random>
 #include <vector>
 constexpr int N         = 4096;   // FFT size
 constexpr int BLOCKSIZE = 128;    // 256 threads / block
 /* ------------------------------------------------------------------ */
 /*                            K E R N E L S                           */
 /* ------------------------------------------------------------------ */
 // 1) GLOBAL-memory baseline ---------------------------------------------------
 __global__ void pw_global_kernel(const hipFloatComplex* __restrict__ X,
                                  const hipFloatComplex* __restrict__ H,
                                  hipFloatComplex*       __restrict__ Y,
                                  int                   n)
 {
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i < n) Y[i] = hipCmulf(X[i], H[i]);
 }
 // 2) SHARED-memory staging ----------------------------------------------------
 __global__ void pw_shared_kernel(const hipFloatComplex* __restrict__ X,
                                  const hipFloatComplex* __restrict__ H,
                                  hipFloatComplex*       __restrict__ Y,
                                  int                   n)
 {
     __shared__ hipFloatComplex Xs[BLOCKSIZE];
     __shared__ hipFloatComplex Hs[BLOCKSIZE];
     int gid = blockIdx.x * blockDim.x + threadIdx.x;
     int tid = threadIdx.x;
     if (gid < n) {
         Xs[tid] = X[gid];
         Hs[tid] = H[gid];
     }
     __syncthreads();
     if (gid < n) Y[gid] = hipCmulf(Xs[tid], Hs[tid]);
 }
 // 3) CONSTANT-memory version --------------------------------------------------
 __constant__ hipFloatComplex H_const[N];
 __global__ void pw_const_kernel(const hipFloatComplex* __restrict__ X,
                                 hipFloatComplex*       __restrict__ Y,
                                 int                   n)
 {
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i < n) Y[i] = hipCmulf(X[i], H_const[i]);
 }
 /* ------------------------------------------------------------------ */
 /*                    S M A L L   L A U N C H E R S                    */
 /* ------------------------------------------------------------------ */
 inline void launch_pw_global(const hipFloatComplex* X,
                              const hipFloatComplex* H,
                              hipFloatComplex*       Y,
                              int                   n,
                              dim3                  grid,
                              dim3                  block,
                              hipStream_t          s = 0)
 {
     void* params[] = { (void*)&X, (void*)&H, (void*)&Y, &n };
     hipLaunchKernel(reinterpret_cast<const void*>((const void*)pw_global_kernel), grid, block,
                      params, 0, s);
 }
 inline void launch_pw_shared(const hipFloatComplex* X,
                              const hipFloatComplex* H,
                              hipFloatComplex*       Y,
                              int                   n,
                              dim3                  grid,
                              dim3                  block,
                              hipStream_t          s = 0)
 {
     void* params[] = { (void*)&X, (void*)&H, (void*)&Y, &n };
     hipLaunchKernel(reinterpret_cast<const void*>((const void*)pw_shared_kernel), grid, block,
                      params, 0, s);
 }
 inline void launch_pw_const (const hipFloatComplex* X,
                              hipFloatComplex*       Y,
                              int                   n,
                              dim3                  grid,
                              dim3                  block,
                              hipStream_t          s = 0)
 {
     void* params[] = { (void*)&X, (void*)&Y, &n };
     hipLaunchKernel(reinterpret_cast<const void*>((const void*)pw_const_kernel), grid, block,
                      params, 0, s);
 }
 /* ------------------------------------------------------------------ */
 /*                              H E L P E R S                         */
 /* ------------------------------------------------------------------ */
 void gpuCheck(hipError_t err, const char* msg)
 {
     if (err != hipSuccess) {
         fprintf(stderr, "CUDA error %s : %s\n", msg, hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 }
 std::vector<hipFloatComplex> randomComplexVector(int n)
 {
     std::mt19937 gen(0xC0FFEE);
     std::uniform_real_distribution<float> dist(-1.f, 1.f);
     std::vector<hipFloatComplex> v(n);
     for (auto& c : v) c = make_hipFloatComplex(dist(gen), dist(gen));
     return v;
 }
//  /i
 // ---- Generic timing helper (1 kernel launch) -----------------------
template<typename Launcher, typename... Args>
float timeKernel(dim3 grid, dim3 block, Launcher launch, Args... args)
{
    hipEvent_t start, stop;
    gpuCheck(hipEventCreate(&start), "create event");
    gpuCheck(hipEventCreate(&stop),  "create event");
    gpuCheck(hipEventRecord(start),  "record start");
    // supply *all* parameters: user args … grid  block  stream(0)
    launch(args..., grid, block, (hipStream_t)0);
    gpuCheck(hipEventRecord(stop),   "record stop");
    gpuCheck(hipEventSynchronize(stop),"sync stop");
    float ms = 0.f;
    gpuCheck(hipEventElapsedTime(&ms, start, stop), "elapsed");
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}
 /* ------------------------------------------------------------------ */
 /*                                M A I N                             */
 /* ------------------------------------------------------------------ */
 int main()
 {
     // 1) Host & device buffers ---------------------------------------
     auto h_X = randomComplexVector(N);
     auto h_H = randomComplexVector(N);
     hipFloatComplex *d_X{}, *d_H{}, *d_Y{};
     size_t bytes = N * sizeof(hipFloatComplex);
     gpuCheck(hipMalloc(&d_X, bytes), "malloc d_X");
     gpuCheck(hipMalloc(&d_H, bytes), "malloc d_H");
     gpuCheck(hipMalloc(&d_Y, bytes), "malloc d_Y");
     gpuCheck(hipMemcpy(d_X, h_X.data(), bytes, hipMemcpyHostToDevice), "cpy X");
     gpuCheck(hipMemcpy(d_H, h_H.data(), bytes, hipMemcpyHostToDevice), "cpy H");
     // 2) Copy H to constant memory -----------------------------------
     gpuCheck(hipMemcpyToSymbol(HIP_SYMBOL(H_const), h_H.data(), bytes), "const copy");
    //  dim3 block(32, 32);   // 32×16 = 512 threads
    //  // Ceil division in each dimension
    // dim3 grid( (N + block.x - 1) / block.x ,
    // (N + block.y - 1) / block.y );
    // // dim3 block(16, 8, 4); // 512 threads in 3-D
    // dim3 grid( (N + block.x - 1) / block.x ,
    // (N + block.y - 1) / block.y );
     dim3 block(BLOCKSIZE);
     dim3 grid((N + BLOCKSIZE - 1) / BLOCKSIZE);
     // 3) Run & time ---------------------------------------------------
     float t_global = timeKernel(grid, block, launch_pw_global,
                                 d_X, d_H, d_Y, N);
     float t_shared = timeKernel(grid, block, launch_pw_shared,
                                 d_X, d_H, d_Y, N);
     float t_const  = timeKernel(grid, block, launch_pw_const,
                                 d_X,       d_Y, N);
     printf("\n==== 4096-point complex point-wise multiply ====\n");
     printf("Global   memory: %7.3f µs\n", t_global * 1000.0f);
     printf("Shared   memory: %7.3f µs\n", t_shared * 1000.0f);
     printf("Constant memory: %7.3f µs\n", t_const  * 1000.0f);
     hipFree(d_X); hipFree(d_H); hipFree(d_Y);
     return 0;
 }
