#include "hip/hip_runtime.h"
// Optimized GPU code for 64000 x 20 x 4096 pointwise multiply using double-buffered streams with detailed timing and CSV logging
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <vector>
#include <random>
#include <chrono>
#include <fstream>

constexpr int RAYS = 64000;
constexpr int FRAMES = 20;
constexpr int N = 4096; // FFT size
constexpr int TOTAL = RAYS * FRAMES * N;
constexpr int BLOCKSIZE = 256;
constexpr int CHUNK = TOTAL / 2; // Use two buffers

__global__ void flat_pw_multiply(const hipFloatComplex* X,
                                 const hipFloatComplex* H,
                                 hipFloatComplex* Y,
                                 int total) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < total) {
        Y[gid] = hipCmulf(X[gid], H[gid]);
    }
}

std::vector<hipFloatComplex> randomComplexVector(int count) {
    std::mt19937 gen(0xC0FFEE);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    std::vector<hipFloatComplex> vec(count);
    for (auto& c : vec) c = make_hipFloatComplex(dist(gen), dist(gen));
    return vec;
}

int main() {
    std::ofstream log("timing_report.csv");
    log << "buffer_id,H2D_ms,kernel_ms,D2H_ms\n";

    size_t chunk_bytes = CHUNK * sizeof(hipFloatComplex);

    hipFloatComplex *h_X[2], *h_H[2], *h_Y[2];
    for (int i = 0; i < 2; ++i) {
        hipHostAlloc(&h_X[i], chunk_bytes, hipHostMallocDefault);
        hipHostAlloc(&h_H[i], chunk_bytes, hipHostMallocDefault);
        hipHostAlloc(&h_Y[i], chunk_bytes, hipHostMallocDefault);
    }

    auto rand_X = randomComplexVector(TOTAL);
    auto rand_H = randomComplexVector(TOTAL);
    memcpy(h_X[0], rand_X.data(), chunk_bytes);
    memcpy(h_H[0], rand_H.data(), chunk_bytes);
    memcpy(h_X[1], rand_X.data() + CHUNK, chunk_bytes);
    memcpy(h_H[1], rand_H.data() + CHUNK, chunk_bytes);

    hipFloatComplex *d_X[2], *d_H[2], *d_Y[2];
    for (int i = 0; i < 2; ++i) {
        hipMalloc(&d_X[i], chunk_bytes);
        hipMalloc(&d_H[i], chunk_bytes);
        hipMalloc(&d_Y[i], chunk_bytes);
    }

    hipStream_t streams[2];
    hipEvent_t start_H2D[2], stop_H2D[2], start_kernel[2], stop_kernel[2], start_D2H[2], stop_D2H[2];
    for (int i = 0; i < 2; ++i) {
        hipStreamCreate(&streams[i]);
        hipEventCreate(&start_H2D[i]);
        hipEventCreate(&stop_H2D[i]);
        hipEventCreate(&start_kernel[i]);
        hipEventCreate(&stop_kernel[i]);
        hipEventCreate(&start_D2H[i]);
        hipEventCreate(&stop_D2H[i]);
    }

    auto wall_start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < 2; ++i) {
        hipEventRecord(start_H2D[i], streams[i]);
        hipMemcpyAsync(d_X[i], h_X[i], chunk_bytes, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_H[i], h_H[i], chunk_bytes, hipMemcpyHostToDevice, streams[i]);
        hipEventRecord(stop_H2D[i], streams[i]);

        hipEventRecord(start_kernel[i], streams[i]);
        dim3 block(BLOCKSIZE);
        dim3 grid((CHUNK + block.x - 1) / block.x);
        flat_pw_multiply<<<grid, block, 0, streams[i]>>>(d_X[i], d_H[i], d_Y[i], CHUNK);
        hipEventRecord(stop_kernel[i], streams[i]);

        hipEventRecord(start_D2H[i], streams[i]);
        hipMemcpyAsync(h_Y[i], d_Y[i], chunk_bytes, hipMemcpyDeviceToHost, streams[i]);
        hipEventRecord(stop_D2H[i], streams[i]);
    }

    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);
    auto wall_end = std::chrono::high_resolution_clock::now();

    float h2d_ms[2], kernel_ms[2], d2h_ms[2];
    for (int i = 0; i < 2; ++i) {
        hipEventElapsedTime(&h2d_ms[i], start_H2D[i], stop_H2D[i]);
        hipEventElapsedTime(&kernel_ms[i], start_kernel[i], stop_kernel[i]);
        hipEventElapsedTime(&d2h_ms[i], start_D2H[i], stop_D2H[i]);
        log << i << "," << h2d_ms[i] << "," << kernel_ms[i] << "," << d2h_ms[i] << "\n";
    }

    double wall_total_us = std::chrono::duration_cast<std::chrono::microseconds>(wall_end - wall_start).count();

    printf("Timing Report (per buffer):\n");
    for (int i = 0; i < 2; ++i) {
        printf("Buffer %d: H2D = %.3f ms, Kernel = %.3f ms, D2H = %.3f ms\n",
               i, h2d_ms[i], kernel_ms[i], d2h_ms[i]);
    }
    printf("Total wall-clock time: %.3f ms\n", wall_total_us / 1000.0);
    log << "Wall, , ," << wall_total_us / 1000.0 << "\n";
    log.close();

    for (int i = 0; i < 2; ++i) {
        hipFree(d_X[i]); hipFree(d_H[i]); hipFree(d_Y[i]);
        hipHostFree(h_X[i]); hipHostFree(h_H[i]); hipHostFree(h_Y[i]);
        hipStreamDestroy(streams[i]);
        hipEventDestroy(start_H2D[i]); hipEventDestroy(stop_H2D[i]);
        hipEventDestroy(start_kernel[i]); hipEventDestroy(stop_kernel[i]);
        hipEventDestroy(start_D2H[i]); hipEventDestroy(stop_D2H[i]);
    }

    return 0;
}
