#include "hip/hip_runtime.h"
// Optimized GPU code for 64000 x 20 x 4096 pointwise multiply using pinned memory and stream
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <vector>
#include <random>
#include <chrono>

constexpr int RAYS = 64000;
constexpr int FRAMES = 20;
constexpr int N = 4096; // FFT size
constexpr int TOTAL = RAYS * FRAMES * N;
constexpr int BLOCKSIZE = 256;

__global__ void flat_pw_multiply(const hipFloatComplex* X,
                                 const hipFloatComplex* H,
                                 hipFloatComplex* Y,
                                 int total) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < total) {
        Y[gid] = hipCmulf(X[gid], H[gid]);
    }
}

std::vector<hipFloatComplex> randomComplexVector(int count) {
    std::mt19937 gen(0xC0FFEE);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    std::vector<hipFloatComplex> vec(count);
    for (auto& c : vec) c = make_hipFloatComplex(dist(gen), dist(gen));
    return vec;
}

int main() {
    size_t bytes = TOTAL * sizeof(hipFloatComplex);

    // Allocate host pinned memory
    hipFloatComplex *h_X_pinned, *h_H_pinned, *h_Y_pinned;
    hipHostAlloc(&h_X_pinned, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_H_pinned, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_Y_pinned, bytes, hipHostMallocDefault);

    // Fill input with dummy data
    auto rand_X = randomComplexVector(TOTAL);
    auto rand_H = randomComplexVector(TOTAL);
    memcpy(h_X_pinned, rand_X.data(), bytes);
    memcpy(h_H_pinned, rand_H.data(), bytes);

    // Allocate device memory
    hipFloatComplex *d_X, *d_H, *d_Y;
    hipMalloc(&d_X, bytes);
    hipMalloc(&d_H, bytes);
    hipMalloc(&d_Y, bytes);

    // Create stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Timing start
    auto start = std::chrono::high_resolution_clock::now();

    // Async H2D copy
    hipMemcpyAsync(d_X, h_X_pinned, bytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_H, h_H_pinned, bytes, hipMemcpyHostToDevice, stream);

    // Kernel launch
    dim3 block(BLOCKSIZE);
    dim3 grid((TOTAL + block.x - 1) / block.x);
    flat_pw_multiply<<<grid, block, 0, stream>>>(d_X, d_H, d_Y, TOTAL);

    // Async D2H copy
    hipMemcpyAsync(h_Y_pinned, d_Y, bytes, hipMemcpyDeviceToHost, stream);

    // Sync
    hipStreamSynchronize(stream);
    auto end = std::chrono::high_resolution_clock::now();

    double total_us = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    printf("Completed 64000x20x4096 in %.2f ms\n", total_us / 1000.0);

    // Cleanup
    hipFree(d_X); hipFree(d_H); hipFree(d_Y);
    hipHostFree(h_X_pinned); hipHostFree(h_H_pinned); hipHostFree(h_Y_pinned);
    hipStreamDestroy(stream);

    return 0;
}
