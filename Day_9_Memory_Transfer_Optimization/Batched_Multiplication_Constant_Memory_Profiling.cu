// Optimized CUDA code using pinned memory and streams for pointwise multiply benchmarking
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <random>
#include <vector>
#include <fstream>

constexpr int N = 4096;
constexpr int BLOCKSIZE = 128;
constexpr int BATCHES = 1000;

__global__ void pw_const_kernel(const hipFloatComplex* __restrict__ X,
                                hipFloatComplex*       __restrict__ Y,
                                int                   n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) Y[i] = hipCmulf(X[i], Y[i]);
}

__constant__ hipFloatComplex H_const[N];

void gpuCheck(hipError_t err, const char* msg)
{
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error %s : %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

std::vector<hipFloatComplex> randomComplexVector(int n)
{
    std::mt19937 gen(0xC0FFEE);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    std::vector<hipFloatComplex> v(n);
    for (auto& c : v) c = make_hipFloatComplex(dist(gen), dist(gen));
    return v;
}

int main()
{
    dim3 block(BLOCKSIZE);
    dim3 grid((N + BLOCKSIZE - 1) / BLOCKSIZE);

    hipFloatComplex *d_X, *d_Y;
    hipFloatComplex *h_X_pinned, *h_Y_pinned;
    size_t bytes = N * sizeof(hipFloatComplex);

    // Allocate pinned host memory
    gpuCheck(hipHostAlloc(&h_X_pinned, bytes, hipHostMallocDefault), "host X pinned");
    gpuCheck(hipHostAlloc(&h_Y_pinned, bytes, hipHostMallocDefault), "host Y pinned");

    // Fill input data
    auto h_X_data = randomComplexVector(N);
    auto h_H_data = randomComplexVector(N);
    memcpy(h_X_pinned, h_X_data.data(), bytes);

    // Allocate device memory
    gpuCheck(hipMalloc(&d_X, bytes), "malloc d_X");
    gpuCheck(hipMalloc(&d_Y, bytes), "malloc d_Y");

    // Copy H to constant memory
    gpuCheck(hipMemcpyToSymbol(HIP_SYMBOL(H_const), h_H_data.data(), bytes), "const copy");

    hipStream_t stream;
    hipStreamCreate(&stream);

    std::ofstream log("stream_timing_log.csv");
    log << "batch_id,copy_time_us,kernel_time_us\n";

    for (int b = 0; b < BATCHES; ++b) {
        hipEvent_t start_copy, stop_copy, start_kernel, stop_kernel;
        hipEventCreate(&start_copy);
        hipEventCreate(&stop_copy);
        hipEventCreate(&start_kernel);
        hipEventCreate(&stop_kernel);

        // Async H2D
        hipEventRecord(start_copy, stream);
        hipMemcpyAsync(d_X, h_X_pinned, bytes, hipMemcpyHostToDevice, stream);
        hipEventRecord(stop_copy, stream);

        // Kernel
        hipEventRecord(start_kernel, stream);
        pw_const_kernel<<<grid, block, 0, stream>>>(d_X, d_Y, N);
        hipEventRecord(stop_kernel, stream);

        // Async D2H
        hipMemcpyAsync(h_Y_pinned, d_Y, bytes, hipMemcpyDeviceToHost, stream);

        // Sync and time
        hipEventSynchronize(stop_kernel);

        float copy_time = 0.f, kernel_time = 0.f;
        hipEventElapsedTime(&copy_time, start_copy, stop_copy);
        hipEventElapsedTime(&kernel_time, start_kernel, stop_kernel);

        log << b << "," << copy_time * 1000.0f << "," << kernel_time * 1000.0f << "\n";

        hipEventDestroy(start_copy);
        hipEventDestroy(stop_copy);
        hipEventDestroy(start_kernel);
        hipEventDestroy(stop_kernel);
    }

    log.close();
    hipFree(d_X); hipFree(d_Y);
    hipHostFree(h_X_pinned); hipHostFree(h_Y_pinned);
    hipStreamDestroy(stream);

    printf("Done. Profile logged in stream_timing_log.csv\n");
    return 0;
}
