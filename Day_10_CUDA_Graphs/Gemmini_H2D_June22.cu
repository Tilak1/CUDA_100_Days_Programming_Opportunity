#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_fp16.h>
#include <random>
#include <vector>
#include <string>
#include <algorithm> // For std::min
#include <nvtx3/nvToolsExt.h>  // NVTX for profiling markers

constexpr int ELEMENTS_PER_SYMBOL = 4096;    // 4K elements per symbol
constexpr int SYMBOLS_PER_RAY = 20;          // 20 symbols per ray
constexpr int TOTAL_RAYS = 2000;             // 1K rays for testing
constexpr int BLOCKSIZE = 128;               // Threads per block
constexpr int STREAMS_PER_GPU = 64;          // Balanced for performance

constexpr int BATCH_SIZE_RAYS = 1000;
constexpr int BATCH_SIZE_SYMBOLS = BATCH_SIZE_RAYS * SYMBOLS_PER_RAY;

using ComplexType = __half2;
const char* precision_name = "Half-Precision (FP16)";

/* ------------------------------------------------------------------ */
/*                            K E R N E L S                           */
/* ------------------------------------------------------------------ */

__global__ void pw_multiply_half_kernel(const __half2* __restrict__ X,
                                        const __half2* __restrict__ H,
                                        __half2*       __restrict__ Y,
                                        int                         n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        __half2 x_val = X[i];
        __half2 h_val = H[i];
        __half x_real = __low2half(x_val);
        __half x_imag = __high2half(x_val);
        __half h_real = __low2half(h_val);
        __half h_imag = __high2half(h_val);
        __half result_real = __hsub(__hmul(x_real, h_real), __hmul(x_imag, h_imag));
        __half result_imag = __hadd(__hmul(x_real, h_imag), __hmul(x_imag, h_real));
        Y[i] = __halves2half2(result_real, result_imag);
    }
}

/* ------------------------------------------------------------------ */
/*                              H E L P E R S                         */
/* ------------------------------------------------------------------ */

void gpuCheck(hipError_t err, const char* msg)
{
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error %s : %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

std::vector<__half2> randomComplexHalfVector(int n, int seed = 0xC0FFEE)
{
    std::mt19937 gen(seed);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    std::vector<__half2> v(n);
    for (auto& c : v) {
        float real = dist(gen);
        float imag = dist(gen);
        c = __halves2half2(__float2half(real), __float2half(imag));
    }
    return v;
}

ComplexType* safeHostAlloc(size_t bytes, const char* name) {
    ComplexType* ptr = nullptr;
    hipError_t err = hipHostAlloc(&ptr, bytes, hipHostMallocDefault);
    if (err == hipSuccess) {
        printf("  %s: hipHostAlloc success (%.2f GB)\n", name, bytes / (1024.0f * 1024.0f * 1024.0f));
        return ptr;
    }
    ptr = (ComplexType*)malloc(bytes);
    if (ptr) {
        printf("  %s: malloc fallback success (%.2f GB)\n", name, bytes / (1024.0f * 1024.0f * 1024.0f));
        return ptr;
    }
    printf("  %s: FAILED to allocate %.2f GB\n", name, bytes / (1024.0f * 1024.0f * 1024.0f));
    return nullptr;
}

void safeFree(ComplexType* ptr) {
    if (ptr) {
        if (hipHostFree(ptr) != hipSuccess) {
            free(ptr);
        }
    }
}

struct TimingResults {
    float h2d_time;
    float kernel_launch_time;
    float kernel_exec_time;
    float d2h_time;
    float total_time;
    int num_kernels_launched;
};

/* ------------------------------------------------------------------ */
/*                                M A I N                             */
/* ------------------------------------------------------------------ */

int main()
{
    nvtxRangePush("Program_Initialization");
    int deviceCount;
    gpuCheck(hipGetDeviceCount(&deviceCount), "get device count");
    if (deviceCount < 1) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    printf("=== 1K Rays Multi-GPU Point-wise Complex Multiplication (%s) ===\n", precision_name);
    printf("Found %d CUDA device(s)\n", deviceCount);
    long long totalOperations = (long long)TOTAL_RAYS * SYMBOLS_PER_RAY * ELEMENTS_PER_SYMBOL;
    printf("Workload: %lld operations (%.1f million)\n", totalOperations, totalOperations / 1000000.0f);
    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t prop;
        gpuCheck(hipGetDeviceProperties(&prop, dev), "get device properties");
        printf("\nDevice %d: %s\n", dev, prop.name);
    }
    printf("\n");
    nvtxRangePop();

    nvtxRangePush("Host_Memory_Allocation_And_Data_Generation");
    size_t batchBytes = (size_t)BATCH_SIZE_SYMBOLS * ELEMENTS_PER_SYMBOL * sizeof(ComplexType);
    printf("Allocating %.2f GB of pinned host memory...\n", (batchBytes * 3) / (1024.0f * 1024.0f * 1024.0f));
    ComplexType *batch_X = safeHostAlloc(batchBytes, "batch_X");
    ComplexType *batch_H = safeHostAlloc(batchBytes, "batch_H");
    ComplexType *batch_Y = safeHostAlloc(batchBytes, "batch_Y");
    if (!batch_X || !batch_H || !batch_Y) return 1;
    printf("Generating test data...\n");
    for (int i = 0; i < BATCH_SIZE_SYMBOLS; i++) {
        auto vec_X = randomComplexHalfVector(ELEMENTS_PER_SYMBOL, i * 2 + 1000);
        auto vec_H = randomComplexHalfVector(ELEMENTS_PER_SYMBOL, i * 2 + 2000);
        memcpy(&batch_X[(size_t)i * ELEMENTS_PER_SYMBOL], vec_X.data(), (size_t)ELEMENTS_PER_SYMBOL * sizeof(ComplexType));
        memcpy(&batch_H[(size_t)i * ELEMENTS_PER_SYMBOL], vec_H.data(), (size_t)ELEMENTS_PER_SYMBOL * sizeof(ComplexType));
    }
    nvtxRangePop();

    // ==============================================
    // SINGLE GPU BASELINE PERFORMANCE
    // ==============================================
    nvtxRangePush("Single_GPU_Performance");
    printf("\n=== SINGLE GPU PERFORMANCE (with CUDA Graph) ===\n");
    
    nvtxRangePush("Single_GPU_Setup");
    ComplexType *dev_X_single, *dev_H_single, *dev_Y_single;
    gpuCheck(hipSetDevice(0), "set device 0");
    gpuCheck(hipMalloc(&dev_X_single, batchBytes), "malloc X single");
    gpuCheck(hipMalloc(&dev_H_single, batchBytes), "malloc H single");
    gpuCheck(hipMalloc(&dev_Y_single, batchBytes), "malloc Y single");
    
    hipStream_t captureStream;
    hipGraph_t graph = nullptr;
    hipGraphExec_t graphExec = nullptr;
    gpuCheck(hipStreamCreate(&captureStream), "create capture stream");
    
    printf("  Phase 1: Capturing and Instantiating CUDA Graph...\n");
    gpuCheck(hipStreamBeginCapture(captureStream, hipStreamCaptureModeGlobal), "begin capture");
    gpuCheck(hipMemcpyAsync(dev_X_single, batch_X, batchBytes, hipMemcpyHostToDevice, captureStream), "capture copy X");
    gpuCheck(hipMemcpyAsync(dev_H_single, batch_H, batchBytes, hipMemcpyHostToDevice, captureStream), "capture copy H");
    dim3 block(BLOCKSIZE);
    int captured_kernels = 0;
    for (int s = 0; s < STREAMS_PER_GPU; s++) {
        int symbolsPerChunk = BATCH_SIZE_SYMBOLS / STREAMS_PER_GPU;
        int remainingSymbols = BATCH_SIZE_SYMBOLS % STREAMS_PER_GPU;
        int chunkStartSymbol = s * symbolsPerChunk;
        int chunkSymbols = symbolsPerChunk + (s == STREAMS_PER_GPU - 1 ? remainingSymbols : 0);
        if (chunkSymbols > 0) {
            int chunkElements = chunkSymbols * ELEMENTS_PER_SYMBOL;
            size_t chunkOffset = (size_t)chunkStartSymbol * ELEMENTS_PER_SYMBOL;
            dim3 grid((chunkElements + BLOCKSIZE - 1) / BLOCKSIZE);
            pw_multiply_half_kernel<<<grid, block, 0, captureStream>>>(
                (ComplexType*)((char*)dev_X_single + chunkOffset), (ComplexType*)((char*)dev_H_single + chunkOffset),
                (ComplexType*)((char*)dev_Y_single + chunkOffset), chunkElements);
            captured_kernels++;
        }
    }
    gpuCheck(hipGetLastError(), "kernel launch capture");
    gpuCheck(hipMemcpyAsync(batch_Y, dev_Y_single, batchBytes, hipMemcpyDeviceToHost, captureStream), "capture copy Y");
    gpuCheck(hipStreamEndCapture(captureStream, &graph), "end capture");
    gpuCheck(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0), "instantiate graph");
    printf("  Graph setup complete for %d kernels.\n", captured_kernels);
    nvtxRangePop(); 

    nvtxRangePush("Single_GPU_Execution_And_Timing");
    printf("  Phase 2: Launching Graph and Synchronizing...\n");
    
    TimingResults singleGpuResults = {0};
    hipEvent_t start_event, stop_event;
    gpuCheck(hipEventCreate(&start_event), "create start");
    gpuCheck(hipEventCreate(&stop_event), "create stop");

    gpuCheck(hipEventRecord(start_event, captureStream), "record start in stream");
    gpuCheck(hipGraphLaunch(graphExec, captureStream), "launch graph");
    gpuCheck(hipEventRecord(stop_event, captureStream), "record stop in stream");
    gpuCheck(hipEventSynchronize(stop_event), "sync stop event");

    gpuCheck(hipEventElapsedTime(&singleGpuResults.total_time, start_event, stop_event), "get total gpu time");
    
    hipEvent_t launch_start_cpu, launch_stop_cpu;
    gpuCheck(hipEventCreateWithFlags(&launch_start_cpu, hipEventBlockingSync), "create launch start");
    gpuCheck(hipEventCreateWithFlags(&launch_stop_cpu, hipEventBlockingSync), "create launch stop");
    gpuCheck(hipEventRecord(launch_start_cpu), "record launch start");
    gpuCheck(hipGraphLaunch(graphExec, captureStream), "launch graph for timing");
    gpuCheck(hipEventRecord(launch_stop_cpu), "record launch stop");
    gpuCheck(hipEventSynchronize(launch_stop_cpu), "sync launch stop event");
    gpuCheck(hipEventElapsedTime(&singleGpuResults.kernel_launch_time, launch_start_cpu, launch_stop_cpu), "get launch time");
    
    singleGpuResults.kernel_exec_time = singleGpuResults.total_time;
    singleGpuResults.num_kernels_launched = 1;

    gpuCheck(hipEventDestroy(start_event), "destroy start");
    gpuCheck(hipEventDestroy(stop_event), "destroy stop");
    gpuCheck(hipEventDestroy(launch_start_cpu), "destroy launch start");
    gpuCheck(hipEventDestroy(launch_stop_cpu), "destroy launch stop");
    nvtxRangePop();
    nvtxRangePop();

    // ==============================================
    // MULTI-GPU PERFORMANCE
    // ==============================================
    nvtxRangePush("Multi_GPU_Performance_Analysis_Mode");
    printf("\n=== MULTI-GPU PERFORMANCE (Serialized Analysis Mode) ===\n");
    
    nvtxRangePush("Multi_GPU_Setup");
    ComplexType **dev_X = new ComplexType*[deviceCount];
    ComplexType **dev_H = new ComplexType*[deviceCount];
    ComplexType **dev_Y = new ComplexType*[deviceCount];
    hipStream_t **streams = new hipStream_t*[deviceCount];
    
    int base = BATCH_SIZE_SYMBOLS / deviceCount;
    int remainder = BATCH_SIZE_SYMBOLS % deviceCount;
    for (int dev = 0; dev < deviceCount; dev++) {
        gpuCheck(hipSetDevice(dev), "set device");
        int gpuSymbols = base + (dev < remainder ? 1 : 0);
        size_t gpuBytes = (size_t)gpuSymbols * ELEMENTS_PER_SYMBOL * sizeof(ComplexType);
        gpuCheck(hipMalloc(&dev_X[dev], gpuBytes), "malloc X");
        gpuCheck(hipMalloc(&dev_H[dev], gpuBytes), "malloc H");
        gpuCheck(hipMalloc(&dev_Y[dev], gpuBytes), "malloc Y");
        streams[dev] = new hipStream_t[STREAMS_PER_GPU];
        for (int s = 0; s < STREAMS_PER_GPU; s++) {
            gpuCheck(hipStreamCreate(&streams[dev][s]), "create stream");
        }
    }
    nvtxRangePop(); 
    
    TimingResults multiGpuResults = {0};
    hipEvent_t phase_start, phase_stop;
    gpuCheck(hipEventCreate(&phase_start), "create phase start");
    gpuCheck(hipEventCreate(&phase_stop), "create phase stop");

    // --- Phase 1: H2D Transfers ---
    nvtxRangePush("Multi_GPU_H2D");
    gpuCheck(hipEventRecord(phase_start, 0), "record h2d start");
    size_t currentHostOffsetSymbols = 0;
    for (int dev = 0; dev < deviceCount; dev++) {
        gpuCheck(hipSetDevice(dev), "set device");
        int gpuSymbols = base + (dev < remainder ? 1 : 0);
        size_t gpuBytes = (size_t)gpuSymbols * ELEMENTS_PER_SYMBOL * sizeof(ComplexType);
        gpuCheck(hipMemcpyAsync(dev_X[dev], &batch_X[currentHostOffsetSymbols * ELEMENTS_PER_SYMBOL], gpuBytes, hipMemcpyHostToDevice, streams[dev][0]), "H2D X");
        gpuCheck(hipMemcpyAsync(dev_H[dev], &batch_H[currentHostOffsetSymbols * ELEMENTS_PER_SYMBOL], gpuBytes, hipMemcpyHostToDevice, streams[dev][0]), "H2D H");
        currentHostOffsetSymbols += gpuSymbols;
    }
    for (int dev = 0; dev < deviceCount; dev++) { gpuCheck(hipSetDevice(dev), "set dev"); gpuCheck(hipStreamSynchronize(streams[dev][0]), "sync h2d stream"); }
    gpuCheck(hipEventRecord(phase_stop, 0), "record h2d stop");
    gpuCheck(hipEventSynchronize(phase_stop), "sync phase stop");
    gpuCheck(hipEventElapsedTime(&multiGpuResults.h2d_time, phase_start, phase_stop), "elapsed h2d");
    nvtxRangePop();

    // --- Phase 2: Kernel Execution ---
    nvtxRangePush("Multi_GPU_Execution");
    gpuCheck(hipEventRecord(phase_start, 0), "record exec start");
    int totalKernelsLaunched = 0;
    for (int dev = 0; dev < deviceCount; dev++) {
        gpuCheck(hipSetDevice(dev), "set device");
        int gpuSymbols = base + (dev < remainder ? 1 : 0);
        int symbolsPerStream = (gpuSymbols + STREAMS_PER_GPU - 1) / STREAMS_PER_GPU;
        for (int s = 0; s < STREAMS_PER_GPU && s * symbolsPerStream < gpuSymbols; s++) {
            int streamStartSymbol = s * symbolsPerStream;
            int streamSymbols = std::min(symbolsPerStream, gpuSymbols - streamStartSymbol);
            if (streamSymbols <= 0) continue;
            int streamElements = streamSymbols * ELEMENTS_PER_SYMBOL;
            size_t streamOffset = (size_t)streamStartSymbol * ELEMENTS_PER_SYMBOL;
            dim3 streamGrid((streamElements + BLOCKSIZE - 1) / BLOCKSIZE);
            pw_multiply_half_kernel<<<streamGrid, block, 0, streams[dev][s]>>>(
                (ComplexType*)((char*)dev_X[dev] + streamOffset), (ComplexType*)((char*)dev_H[dev] + streamOffset),
                (ComplexType*)((char*)dev_Y[dev] + streamOffset), streamElements);
            totalKernelsLaunched++;
        }
    }
    for (int dev = 0; dev < deviceCount; dev++) { gpuCheck(hipSetDevice(dev), "set dev"); for (int s = 0; s < STREAMS_PER_GPU; s++) gpuCheck(hipStreamSynchronize(streams[dev][s]), "sync kernel stream");}
    gpuCheck(hipEventRecord(phase_stop, 0), "record exec stop");
    gpuCheck(hipEventSynchronize(phase_stop), "sync phase stop");
    gpuCheck(hipEventElapsedTime(&multiGpuResults.kernel_exec_time, phase_start, phase_stop), "elapsed exec");
    multiGpuResults.num_kernels_launched = totalKernelsLaunched;
    nvtxRangePop();

    // --- Phase 3: D2H Transfers ---
    nvtxRangePush("Multi_GPU_D2H");
    gpuCheck(hipEventRecord(phase_start, 0), "record d2h start");
    currentHostOffsetSymbols = 0;
    for (int dev = 0; dev < deviceCount; dev++) {
        gpuCheck(hipSetDevice(dev), "set device");
        int gpuSymbols = base + (dev < remainder ? 1 : 0);
        size_t gpuBytes = (size_t)gpuSymbols * ELEMENTS_PER_SYMBOL * sizeof(ComplexType);
        gpuCheck(hipMemcpyAsync(&batch_Y[currentHostOffsetSymbols * ELEMENTS_PER_SYMBOL], dev_Y[dev], gpuBytes, hipMemcpyDeviceToHost, streams[dev][0]), "D2H Y");
        currentHostOffsetSymbols += gpuSymbols;
    }
    for (int dev = 0; dev < deviceCount; dev++) { gpuCheck(hipSetDevice(dev), "set dev"); gpuCheck(hipStreamSynchronize(streams[dev][0]), "sync d2h stream");}
    gpuCheck(hipEventRecord(phase_stop, 0), "record d2h stop");
    gpuCheck(hipEventSynchronize(phase_stop), "sync phase stop");
    gpuCheck(hipEventElapsedTime(&multiGpuResults.d2h_time, phase_start, phase_stop), "elapsed d2h");
    nvtxRangePop();
    
    multiGpuResults.total_time = multiGpuResults.h2d_time + multiGpuResults.kernel_exec_time + multiGpuResults.d2h_time;

    gpuCheck(hipEventDestroy(phase_start), "destroy phase start");
    gpuCheck(hipEventDestroy(phase_stop), "destroy phase stop");
    nvtxRangePop();


    // ==============================================
    // DETAILED PERFORMANCE COMPARISON
    // ==============================================
    nvtxRangePush("Performance_Analysis");
    printf("\n=== DETAILED PERFORMANCE COMPARISON ===\n");
    printf("\n📊 DETAILED TIMING BREAKDOWN:\n");
    printf("                          Single GPU    Multi-GPU (%d)    Speedup\n", deviceCount);
    printf("                          -----------    --------------    -------\n");
    printf("H2D Transfer:           %8s      %8.2f ms      %s\n", "N/A*", multiGpuResults.h2d_time, "N/A");
    
    float multiLaunchTime = multiGpuResults.total_time - multiGpuResults.h2d_time - multiGpuResults.kernel_exec_time - multiGpuResults.d2h_time;
    multiLaunchTime = std::max(0.0f, multiLaunchTime); 
    float launchSpeedup = (multiLaunchTime > 0.0001f) ? (singleGpuResults.kernel_launch_time / multiLaunchTime) : 0.0f;
    float execSpeedup = (multiGpuResults.kernel_exec_time > 0.0001f) ? (singleGpuResults.kernel_exec_time / multiGpuResults.kernel_exec_time) : 0.0f;
    float totalSpeedup = (multiGpuResults.total_time > 0.0001f) ? (singleGpuResults.total_time / multiGpuResults.total_time) : 0.0f;
    
    printf("Kernel/Graph Launch:    %8.3f ms      %8.3f ms      %6.2fx\n", singleGpuResults.kernel_launch_time, multiLaunchTime, launchSpeedup);
    printf("Kernel/Graph Execution: %8.2f ms      %8.2f ms      %6.2fx\n", singleGpuResults.kernel_exec_time, multiGpuResults.kernel_exec_time, execSpeedup);
    printf("D2H Transfer:           %8s      %8.2f ms      %s\n", "N/A*", multiGpuResults.d2h_time, "N/A");
    printf("---------------------     -----------    --------------    -------\n");
    printf("TOTAL TIME:             %8.2f ms      %8.2f ms      %6.2fx\n", singleGpuResults.total_time, multiGpuResults.total_time, totalSpeedup);
    printf("*N/A: For graphs, transfers are part of total execution time.\n");
    
    printf("\n📈 PERFORMANCE METRICS:\n");
    printf("Single GPU (with CUDA Graph):\n");
    printf("  Graph launches:         %d (from %d kernels)\n", singleGpuResults.num_kernels_launched, captured_kernels);
    printf("  Launch overhead/graph:  %.2f μs\n", singleGpuResults.kernel_launch_time * 1000.0f);
    printf("  Complex ops/second:     %.1f billion ops/s\n", (float)totalOperations / (singleGpuResults.total_time / 1000.0f) / 1000000000.0f);
    
    printf("\nMulti-GPU (%d devices):\n");
    printf("  Kernels launched:       %d\n", multiGpuResults.num_kernels_launched);
    printf("  Complex ops/second:     %.1f billion ops/s\n", (float)totalOperations / (multiGpuResults.total_time / 1000.0f) / 1000000000.0f);
    
    printf("\n🚀 EFFICIENCY ANALYSIS:\n");
    printf("Total Time Speedup: %.2fx\n", totalSpeedup);
    printf("Parallel Efficiency: %.1f%% (%.1f%% is ideal)\n", (totalSpeedup / deviceCount) * 100.0f, 100.0f);
    nvtxRangePop();
    
    // Cleanup
    nvtxRangePush("Cleanup");
    printf("\nCleaning up memory...\n");
    safeFree(batch_X); safeFree(batch_H); safeFree(batch_Y);
    gpuCheck(hipSetDevice(0), "set device 0");
    gpuCheck(hipFree(dev_X_single), "free X single");
    gpuCheck(hipFree(dev_H_single), "free H single");
    gpuCheck(hipFree(dev_Y_single), "free Y single");
    if (graphExec != nullptr) gpuCheck(hipGraphExecDestroy(graphExec), "destroy graph exec");
    if (graph != nullptr) gpuCheck(hipGraphDestroy(graph), "destroy graph");
    gpuCheck(hipStreamDestroy(captureStream), "destroy capture stream");
    for (int dev = 0; dev < deviceCount; dev++) {
        gpuCheck(hipSetDevice(dev), "set device");
        for (int s = 0; s < STREAMS_PER_GPU; s++) gpuCheck(hipStreamDestroy(streams[dev][s]), "destroy stream");
        delete[] streams[dev];
        gpuCheck(hipFree(dev_X[dev]), "free X");
        gpuCheck(hipFree(dev_H[dev]), "free H");
        gpuCheck(hipFree(dev_Y[dev]), "free Y");
    }
    delete[] dev_X; delete[] dev_H; delete[] dev_Y; delete[] streams;
    nvtxRangePop(); 
    
    printf("\n1K rays processing and analysis complete!\n");
    return 0;
}